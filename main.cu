#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define NUM_HIDDEN_LAYERS 2
#define TILE_k 32
#define MAX_IMAGESIZE 784  // 28x28 flattened size
#define NUM_TRAIN 60000
#define NUM_TEST 10000
#define SIZE 784
#define LR 0.001
#define TRAIN_IMAGE "D:/gpu_project_course/fashion/train-images-idx3-ubyte"
#define TRAIN_LABEL "D:/gpu_project_course/fashion/train-labels-idx1-ubyte"
#define TEST_IMAGE "D:/gpu_project_course/fashion/t10k-images-idx3-ubyte"
#define TEST_LABEL "D:/gpu_project_course/fashion/t10k-labels-idx1-ubyte"
#define HIDDEN_SIZE 128


#define CHECK(call)\
{\
    hipError_t errorSync = call;\
    hipError_t errorASync = hipPeekAtLastError();\
    if (errorSync != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", errorSync,\
                hipGetErrorString(errorSync));\
        exit(EXIT_FAILURE);\
    }\
    if (errorASync != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", errorASync,\
                hipGetErrorString(errorASync));\
        exit(EXIT_FAILURE);\
    }\
}

// Allocate memory for a matrix
float* allocMatrix(int rowSize, int colSize = 1) {
    return (float*)malloc(rowSize * colSize * sizeof(float));
}

// Free allocated memory
void freeMatrix(float* matrix) {
    if (matrix) {
        free(matrix);
    }
}

// Initialize a random matrix
float* initRandomMatrix(int rowSize, int colSize = 1, float lower = 0.0, float upper = 1.0) {
    int size = rowSize * colSize;
    float* res = allocMatrix(rowSize, colSize);
    for (int i = 0; i < size; i++) {
        res[i] = lower + static_cast<float>(rand()) / (static_cast<float>(RAND_MAX / (upper - lower)));
    }
    return res;
}

// Copy values from one matrix to another
void copyMatrix(float* dest, float* src, int size) {
    for (int i = 0; i < size; i++) {
        dest[i] = src[i];
    }
}

// Get label for a given index
const char* getLabelByIdx(int idx) {
    switch (idx) {
        case 0: return "T-shirt/top";
        case 1: return "Trouser";
        case 2: return "Pullover";
        case 3: return "Dress";
        case 4: return "Coat";
        case 5: return "Sandal";
        case 6: return "Shirt";
        case 7: return "Sneaker";
        case 8: return "Bag";
        case 9: return "Ankle boot";
        default: return "Not exist label";
    }
}
// Reverse integer bytes for MNIST file format
int reverseInt(int i) {
    unsigned char c1, c2, c3, c4;
    c1 = i & 255;
    c2 = (i >> 8) & 255;
    c3 = (i >> 16) & 255;
    c4 = (i >> 24) & 255;
    return ((int)c1 << 24) + ((int)c2 << 16) + ((int)c3 << 8) + c4;
}

float* readLabels(const char* path, int* num_labels) {
    FILE* file = fopen(path, "rb");
    if (file == NULL) {
        perror("Error opening file");
        return nullptr;
    }

    int magic_number = 0;
    fread(&magic_number, sizeof(magic_number), 1, file);
    magic_number = reverseInt(magic_number);

    fread(num_labels, sizeof(*num_labels), 1, file);
    *num_labels = reverseInt(*num_labels);

    if (magic_number != 2049) {
        printf("Invalid magic number: %d. Expected 2049 for label file.\n", magic_number);
        fclose(file);
        return nullptr;
    }

    float* labels = (float*)malloc((*num_labels) * sizeof(float));
    for (int i = 0; i < *num_labels; ++i) {
        unsigned char temp = 0;
        fread(&temp, sizeof(temp), 1, file);
        labels[i] = (float)temp;
    }

    fclose(file);
    return labels;
}

float* readImages(const char* path, int* num_images, int* image_size) {
    FILE* file = fopen(path, "rb");
    if (file == NULL) {
        perror("Error opening file");
        return nullptr;
    }

    int magic_number = 0, n_rows = 0, n_cols = 0;
    fread(&magic_number, sizeof(magic_number), 1, file);
    magic_number = reverseInt(magic_number);

    fread(num_images, sizeof(*num_images), 1, file);
    *num_images = reverseInt(*num_images);

    fread(&n_rows, sizeof(n_rows), 1, file);
    n_rows = reverseInt(n_rows);

    fread(&n_cols, sizeof(n_cols), 1, file);
    n_cols = reverseInt(n_cols);

    if (magic_number != 2051 || n_rows != 28 || n_cols != 28) {
        printf("Invalid file format or dimensions. MNIST expects 28x28 images.\n");
        fclose(file);
        return nullptr;
    }

    *image_size = n_rows * n_cols;
    float* images = (float*)malloc((*num_images) * (*image_size) * sizeof(float));

    for (int i = 0; i < *num_images; ++i) {
        for (int j = 0; j < *image_size; ++j) {
            unsigned char temp = 0;
            fread(&temp, sizeof(temp), 1, file);
            images[i * (*image_size) + j] = (float)temp / 255.0f;  // Normalize to [0, 1]
        }
    }

    fclose(file);
    return images;
}

void displayImg(const float* image, int rows, int cols) {
    for (int r = 0; r < rows; ++r) {
        for (int c = 0; c < cols; ++c) {
            if (image[r * cols + c] > 0.0f) {
                printf("* ");
            } else {
                printf("  ");
            }
        }
        printf("\n");
    }
}


// Apply ReLU activation
void applyRelu(float* data, int size) {
    for (int i = 0; i < size; i++) {
        data[i] = data[i] > 0 ? data[i] : 0;
    }
}

// Compute softmax activation
void softmax(float* input, int size) {
    float sum = 0.0f;
    for (int i = 0; i < size; i++) {
        input[i] = expf(input[i]);
        sum += input[i];
    }
    for (int i = 0; i < size; i++) {
        input[i] /= sum;
    }
}

// Compute loss (cross-entropy)
float computeLoss(float* output, int label, int size) {
    float loss = 0.0f;
    float *labels = (float*) malloc(sizeof(float)* size);
    for(int i =0;i< size;i++) labels[i] = 0.0;
    labels[label] = 1.0;
    for (int i = 0; i < size; i++) {
        loss -= labels[i] * logf(output[i]);
    }
    free(labels);
    return loss;
}

void applyActivationDerivative(float* gradient, float* activation, int size) {
    for (int i = 0; i < size; i++) {
        gradient[i] *= (activation[i] > 0) ? 1 : 0;
    }
}

/*
    n is batch size
    d is 728
    Input size : n x d 
    

*/


__global__ void matrixMultiKernel(float* A, float* B, float* C, int m, int n, int k) {
    __shared__ float s_A[TILE_k][TILE_k];
    __shared__ float s_B[TILE_k][TILE_k];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float s = 0.0f;

    for (int batch_idx = 0; batch_idx < (n + TILE_k - 1) / TILE_k; batch_idx++) {
        int A_col = batch_idx * TILE_k + threadIdx.x;
        int B_row = batch_idx * TILE_k + threadIdx.y;

        // Load tiles into shared memory
        s_A[threadIdx.y][threadIdx.x] = (row < m && A_col < n) ? A[row * n + A_col] : 0.0f;
        s_B[threadIdx.y][threadIdx.x] = (col < k && B_row < n) ? B[B_row * k + col] : 0.0f;

        __syncthreads();

        // Perform partial matrix multiplication
        for (int i = 0; i < TILE_k; i++) {
            s += s_A[threadIdx.y][i] * s_B[i][threadIdx.x];
        }

        __syncthreads();
    }

    // Write the result to the output matrix
    if (row < m && col < k) {
        C[row * k + col] = s;
    }
}

// Matrix multiplication wrapper
void matrixMultiplication(float* A, int m, int n, float* B, int k, float* C, bool useDevice = false, dim3 blockSize = dim3(1)) {
    if (!useDevice) {
        for (int i = 0; i < m; i++) {
            for (int j = 0; j < k; j++) {
                float sum = 0.0f;
                for (int t = 0; t < n; t++) {
                    sum += A[i * n + t] * B[t * k + j];
                }
                C[i * k + j] = sum;
            }
        }
    } else {
        float *d_A, *d_B, *d_C;
        hipMalloc((void**)&d_A, m * n * sizeof(float));
        hipMalloc((void**)&d_B, n * k * sizeof(float));
        hipMalloc((void**)&d_C, m * k * sizeof(float));

        hipMemcpy(d_A, A, m * n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, n * k * sizeof(float), hipMemcpyHostToDevice);

        dim3 gridSize((k + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);
        matrixMultiKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, m, n, k);

        hipMemcpy(C, d_C, m * k * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }
}

float* transpose(float* matrix, int rowSize, int colSize, bool useDevice=false){
    float *output = initRandomMatrix(colSize, rowSize);
    
    if(!useDevice){
        for(int i=0;i < rowSize;i++){
            for(int j=0;j < colSize; j++){
                output[j * rowSize + i] = matrix[i * rowSize + j];
            }
        }
    }
    
    return output;
}

void computeGradientForOutputLayer(float* output, float* gradOutput, float* targetLabels, int sampleSize, int outputSize=10, bool useDevice=false) {
    if (!useDevice) {
        for (int i = 0; i < sampleSize; i++) {
            for (int j = 0; j < outputSize; j++) {
                gradOutput[i * outputSize + j] = output[i * outputSize + j];
            }
            gradOutput[i * outputSize + (int)targetLabels[i]] -= 1.0;
        }
    }
}

void computeGradientForOutputBias(float* gradOutput, float* gradBias, int sampleSize, int outputSize=10, bool useDevice=false) {
    if (!useDevice) {
        for (int j = 0; j < outputSize; j++) {
            gradBias[j] = 0.0;
            for (int i = 0; i < sampleSize; i++) {
                gradBias[j] += gradOutput[i * outputSize + j];
            }
        }
    }
}


float computeDerivativeHiddenLayer(float &a) {
    return a > 0 ? a : 0;
}

float multiply(float& a, float& b) {
    return a * b;
}

void elementWiseUnary(float* a, float* c, int rowSize, int colSize, float (*unary)(float&), bool useDevice=false) {
    if (!useDevice) {
        for (int i = 0; i < rowSize; i++) {
            for (int j = 0; j < colSize; j++) {
                c[i * colSize + j] = unary(a[i * colSize + j]);
            }
        }
    }
}

void elementWiseBinary(float* a, float* b, float* c, int rowSize, int colSize, float (*binary)(float&, float&), bool useDevice=false) {
    if (!useDevice) {
        for (int i = 0; i < rowSize; i++) {
            for (int j = 0; j < colSize; j++) {
                c[i * colSize + j] = binary(a[i * colSize + j], b[i * colSize + j]);
            }
        }
    }
}

float addition(float& a, float& b){
    return a + b;
}

void forward(float* input, float** hiddenWeights, float** activations, float** bias, float* output, int outputSize, int sampleSize, int featureSize=728, bool useDevice = false, dim3 blockSize = dim3(1)) {
    float* currentInput = input;
    int currentInputSize = featureSize;
    
    for (int i = 0; i < NUM_HIDDEN_LAYERS - 1; i++) {
        matrixMultiplication(currentInput, sampleSize, currentInputSize, hiddenWeights[i], HIDDEN_SIZE, activations[i], useDevice, blockSize);
        elementWiseBinary(activations[i], bias[i], activations[i],sampleSize, HIDDEN_SIZE,  addition);
        applyRelu(activations[i], HIDDEN_SIZE);
        
        currentInputSize = HIDDEN_SIZE;
        currentInput = activations[i];
    }
    
    matrixMultiplication(currentInput, 1, HIDDEN_SIZE, 
                         hiddenWeights[NUM_HIDDEN_LAYERS-1], outputSize, 
                         output, useDevice, blockSize);
    
    softmax(output, outputSize);
}


float updateWeight(float& org, float& grad) {
    return org - LR * grad;
}


void backward(float* input, float* output, float* targetLabels, float** hiddenWeights, float** activations, float** bias, int sampleSize, int featureSize = 728, int outputSize = 10) {
    float* gradOutput = (float*)malloc(sampleSize * outputSize * sizeof(float));
    float** gradWeights = (float**)malloc(NUM_HIDDEN_LAYERS * sizeof(float*));
    float** gradBias = (float**)malloc(NUM_HIDDEN_LAYERS * sizeof(float*));
    
    for (int i = 0; i < NUM_HIDDEN_LAYERS; i++) {
        int rowSize = (i == 0) ? featureSize : HIDDEN_SIZE;
        int colSize = (i == NUM_HIDDEN_LAYERS - 1) ? outputSize : HIDDEN_SIZE;
        gradWeights[i] = (float*)malloc(rowSize * colSize * sizeof(float));
        gradBias[i] = (float*)malloc(colSize * sizeof(float));
    }
    
    computeGradientForOutputLayer(output, gradOutput, targetLabels, sampleSize, outputSize);

    float* gradientToLoss = gradOutput;

    for (int i = NUM_HIDDEN_LAYERS - 1; i >= 0; i--) {
        int activationColSize = (i == 0) ? featureSize : HIDDEN_SIZE;
        int weightColSize = (i == NUM_HIDDEN_LAYERS - 1) ? outputSize : HIDDEN_SIZE;
        int weightRowSize = (i == 0) ? featureSize : HIDDEN_SIZE;

        float* activationsTransposed = transpose(activations[i], sampleSize, activationColSize);
        
        matrixMultiplication(activationsTransposed, activationColSize, sampleSize, gradientToLoss, weightColSize, gradWeights[i]);
        
        computeGradientForOutputBias(gradientToLoss, gradBias[i], sampleSize, weightColSize);
        
        // If not the first layer, propagate gradient backwards
        if (i > 0) {
            float* weightsTransposed = transpose(hiddenWeights[i], weightRowSize, weightColSize);
            float* derivativeOfActivation = (float*)malloc(sampleSize * activationColSize * sizeof(float));
            
            float* previousGradient = (float*)malloc(sampleSize * activationColSize * sizeof(float));
            matrixMultiplication(gradientToLoss, sampleSize, weightColSize, weightsTransposed, activationColSize, previousGradient);

            elementWiseUnary(activations[i - 1], derivativeOfActivation, sampleSize, activationColSize, computeDerivativeHiddenLayer);
            elementWiseBinary(previousGradient, derivativeOfActivation, previousGradient, sampleSize, activationColSize, multiply);

            gradientToLoss = previousGradient;

            free(weightsTransposed);
            free(derivativeOfActivation);
        }
        
        free(activationsTransposed);
    }
    
    //Update weights && bias
    for (int i = NUM_HIDDEN_LAYERS - 1; i >= 0; i--) {
        int activationColSize = (i == 0) ? featureSize : HIDDEN_SIZE;
        int weightColSize = (i == NUM_HIDDEN_LAYERS - 1) ? outputSize : HIDDEN_SIZE;
        int weightRowSize = (i == 0) ? featureSize : HIDDEN_SIZE;
        elementWiseBinary(hiddenWeights[i], gradWeights[i], hiddenWeights[i], weightRowSize, weightColSize, updateWeight);
        elementWiseBinary(bias[i], gradBias[i], bias[i], sampleSize, activationColSize, updateWeight);
    }
    
    for (int i = 0; i < NUM_HIDDEN_LAYERS; i++) {
        free(gradWeights[i]);
        free(gradBias[i]);
    }

    free(gradWeights);
    free(gradBias);
    free(gradOutput);
}




void train(float** dataset, float* labels, int epochSize, int sampleSize, int inputSize, int outputSize=10) {
    float** hiddenWeights = (float**) malloc(NUM_HIDDEN_LAYERS * sizeof(float*));
    float** activations = (float**) malloc(NUM_HIDDEN_LAYERS * sizeof(float*));
    float** bias = (float**) malloc(NUM_HIDDEN_LAYERS * sizeof(float*));
    for (int i = 0; i < NUM_HIDDEN_LAYERS; i++) {
        int prevSize = (i == 0) ? inputSize : HIDDEN_SIZE;
        int currSize = (i == NUM_HIDDEN_LAYERS - 1) ? outputSize : HIDDEN_SIZE;
        
        hiddenWeights[i] = (float*) malloc(prevSize * currSize * sizeof(float));
        activations[i] = (float*) malloc(currSize * sizeof(float));
        bias[i] = (float*) malloc(currSize * sizeof(float)); 
        
        for (int j = 0; j < prevSize * currSize; j++) {
            hiddenWeights[i][j] = (float)rand() / RAND_MAX - 0.5f;
        }
    }
    printf("Finish initialize weights and bias\n");
    // Training loop
    for (int epoch = 0; epoch < epochSize; epoch++) {
        float totalLoss = 0.0f;
        for (int sampleIdx = 0; sampleIdx < sampleSize; sampleIdx++) {
            float* sample = dataset[sampleIdx];
            float* output = (float*) malloc(outputSize * sizeof(float) * sampleSize);
            forward(sample, hiddenWeights, activations, bias, output, outputSize, sampleSize);
            backward(sample, output, &labels[sampleIdx], hiddenWeights, activations, bias, sampleSize);
            
            free(output);
        }
        printf("Epoch %d, Loss: %.4f\n", epoch + 1, totalLoss / sampleSize);

    }

    // Memory cleanup
    for (int i = 0; i < NUM_HIDDEN_LAYERS; i++) {
        free(hiddenWeights[i]);
        free(activations[i]);
    }
    free(hiddenWeights);
    free(activations);
}

int main() {
    int train_image_count, train_label_count;
    int image_size;
    float* train_images = readImages(TRAIN_IMAGE, &train_image_count, &image_size);
    float* train_labels = readLabels(TRAIN_LABEL, &train_label_count);
    if (!train_images || !train_labels) {
        printf("Failed to load MNIST data.\n");
        return 1;
    }

    const int outputSize = 10;
    const int epochs = 100;

    // Prepare dataset as 2D array of pointers
    float** dataset = (float**) malloc(train_image_count * sizeof(float*));
    for (int i = 0; i < train_image_count; i++) {
        dataset[i] = train_images + i * image_size;
    }
    printf("Loaded Fashion MNIST data.\n");
    // Train the neural network

    printf("Trainning start\n");
    train(dataset, train_labels, epochs, train_image_count, image_size, outputSize);
    printf("Trainning end\n");

    // Cleanup
    free(dataset);
    free(train_labels);
    free(train_images);
    free(train_labels);
    
    return 0;
}
